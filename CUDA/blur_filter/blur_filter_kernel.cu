#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void blur_filter_kernel (const float *in, float *out, int size)
{

    int curr_row, curr_col;
 	int row = blockIdx.y * blockDim.y + threadIdx.y;   /* Obtain row number of pixel */
 	int col = blockIdx.x * blockDim.x + threadIdx.x;   /* Obtain column number of pixel */

    /* Apply blur filter to current pixel */
    if ((row < size) && (col < size)) {
    	float blur_value = 0.0;
    	int num_neighbors = 0;
    	for (int i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++) {
        	for (int j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++) {
                /* Accumulate values of neighbors while checking for
                /* boundary conditions */
                curr_row = row + i;
                curr_col = col + j;
                if ((curr_row > -1) && (curr_row < size) && (curr_col > -1) && (curr_col < size)) {
                        blur_value += in[curr_row * size + curr_col];
                        num_neighbors++;
                    }
                }
            }

    	/* Write averaged blurred value out */
        out[row * size + col] = blur_value / num_neighbors;
    }
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
